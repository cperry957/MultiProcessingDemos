#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef BLOCKSIZE
#define BLOCKSIZE		32		// number of threads per block
#endif

#ifndef SIZE
#define SIZE			1*1024*1024	// array size
#endif

#ifndef NUMTRIALS
#define NUMTRIALS		100		// to make the timing more accurate
#endif

#ifndef TOLERANCE
#define TOLERANCE		0.00001f	// tolerance to relative error
#endif

// ranges for the random numbers:
const float XCMIN = -1.0;
const float XCMAX = 1.0;
const float YCMIN = 0.0;
const float YCMAX = 2.0;
const float RMIN = 0.5;
const float RMAX = 2.0;

// function prototypes:
float		Ranf(float, float);
int		Ranf(int, int);
void		TimeOfDaySeed();

// Monte Carlo (CUDA Kernel) on the device:

__global__  void MonteCarlo(float* A, float* B, float* C, float* D)
{
	__shared__ float prods[BLOCKSIZE];

	unsigned int numItems = blockDim.x;
	unsigned int tnum = threadIdx.x;
	unsigned int wgNum = blockIdx.x;
	unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;

	//prods[tnum] = A[gid] * B[gid];
	prods[tnum] = 1.;

	for (int offset = 1; offset < numItems; offset *= 2)
	{
		int mask = 2 * offset - 1;
		__syncthreads();
		if ((tnum & mask) == 0)
		{
			prods[tnum] += prods[tnum + offset];
		}
	}

	__syncthreads();
	if (tnum == 0)
		D[wgNum] = prods[0];
}

// main program:

int main(int argc, char* argv[])
{
	int dev = findCudaDevice(argc, (const char**)argv);
	TimeOfDaySeed();		// seed the random number generator

	// allocate host memory:

	float* xcs = new float[SIZE];
	float* ycs = new float[SIZE];
	float* rs = new float[SIZE];

	float* hits = new float[SIZE / BLOCKSIZE];

	// fill the random-value arrays:
	for (int i = 0; i < SIZE; i++)
	{
		xcs[i] = Ranf(XCMIN, XCMAX);
		ycs[i] = Ranf(YCMIN, YCMAX);
		rs[i] = Ranf(RMIN, RMAX);
	}

	// get ready to record the maximum performance and the probability:
	float maxPerformance = 0.;      // must be declared outside the NUMTRIES loop
	float currentProb;              // must be declared outside the NUMTRIES loop

	// allocate device memory:

	float* dA, * dB, * dC, * dD;

	dim3 dimsA(SIZE, 1, 1);
	dim3 dimsB(SIZE, 1, 1);
	dim3 dimsC(SIZE, 1, 1);
	dim3 dimsD(SIZE / BLOCKSIZE, 1, 1);

	//__shared__ float prods[SIZE/BLOCKSIZE];


	hipError_t status;
	status = hipMalloc(reinterpret_cast<void**>(&dA), SIZE * sizeof(float));
	checkCudaErrors(status);
	status = hipMalloc(reinterpret_cast<void**>(&dB), SIZE * sizeof(float));
	checkCudaErrors(status);
	status = hipMalloc(reinterpret_cast<void**>(&dC), SIZE * sizeof(float));
	checkCudaErrors(status);
	status = hipMalloc(reinterpret_cast<void**>(&dD), (SIZE / BLOCKSIZE) * sizeof(float));
	checkCudaErrors(status);


	// copy host memory to the device:

	status = hipMemcpy(dA, xcs, SIZE * sizeof(float), hipMemcpyHostToDevice);
	checkCudaErrors(status);
	status = hipMemcpy(dB, ycs, SIZE * sizeof(float), hipMemcpyHostToDevice);
	checkCudaErrors(status);
	status = hipMemcpy(dC, rs, SIZE * sizeof(float), hipMemcpyHostToDevice);
	checkCudaErrors(status);

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1);
	dim3 grid(SIZE / threads.x, 1, 1);

	// Create and start timer

	hipDeviceSynchronize();

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate(&start);
	checkCudaErrors(status);
	status = hipEventCreate(&stop);
	checkCudaErrors(status);

	// record the start event:

	status = hipEventRecord(start, NULL);
	checkCudaErrors(status);

	// execute the kernel:

	for (int t = 0; t < NUMTRIALS; t++)
	{
		MonteCarlo << < grid, threads >> > (dA, dB, dC, dD);
	}

	// record the stop event:

	status = hipEventRecord(stop, NULL);
	checkCudaErrors(status);

	// wait for the stop event to complete:

	status = hipEventSynchronize(stop);
	checkCudaErrors(status);

	float msecTotal = 0.0f;
	status = hipEventElapsedTime(&msecTotal, start, stop);
	checkCudaErrors(status);

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	double multsPerSecond = (float)SIZE * (float)NUMTRIALS / secondsTotal;
	double megaMultsPerSecond = multsPerSecond / 1000000.;
	fprintf(stderr, "Array Size = %10d, MegaMultReductions/Second = %10.2lf\n", SIZE, megaMultsPerSecond);

	// copy result from the device to the host:

	status = hipMemcpy(hits, dD, (SIZE / BLOCKSIZE) * sizeof(float), hipMemcpyDeviceToHost);
	checkCudaErrors(status);

	// check the sum :

	double sum = 0.;
	for (int i = 0; i < SIZE / BLOCKSIZE; i++)
	{
		//fprintf(stderr, "hC[%6d] = %10.2f\n", i, hC[i]);
		sum += (double)hits[i];
	}
	fprintf(stderr, "\nsum = %10.2lf\n", sum);

	// clean up memory:
	delete[] xcs;
	delete[] ycs;
	delete[] rs;
	delete[] hits;

	status = hipFree(dA);
	checkCudaErrors(status);
	status = hipFree(dB);
	checkCudaErrors(status);
	status = hipFree(dC);
	checkCudaErrors(status);
	status = hipFree(dD);
	checkCudaErrors(status);


	return 0;
}

float Ranf(float low, float high)
{
	float r = (float)rand();               // 0 - RAND_MAX
	float t = r / (float)RAND_MAX;       // 0. - 1.

	return   low + t * (high - low);
}

int
Ranf(int ilow, int ihigh)
{
	float low = (float)ilow;
	float high = ceil((float)ihigh);

	return (int)Ranf(low, high);
}

void TimeOfDaySeed()
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time(&timer);
	double seconds = difftime(timer, mktime(&y2k));
	unsigned int seed = (unsigned int)(1000. * seconds);    // milliseconds
	srand(seed);
}